#include "hip/hip_runtime.h"
/**
 * 3mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Problem size. */
// # define ni 512
// # define nj 512
// # define nk 512
// # define nl 512
// # define nm 512

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void init_array(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, int ni, int nj, int nk, int nl, int nm)
{
	int i, j;

	for (i = 0; i < ni; i++)
	{
		for (j = 0; j < nk; j++)
		{
			A[i*nk + j] = ((DATA_TYPE) i*j) / ni;
		}
	}
  
	for (i = 0; i < nk; i++)
	{
		for (j = 0; j < nj; j++)
		{
			B[i*nj + j] = ((DATA_TYPE) i*(j+1)) / nj;
		}
	}
  
	for (i = 0; i < nj; i++)
	{
		for (j = 0; j < nm; j++)
		{
			C[i*nm + j] = ((DATA_TYPE) i*(j+3)) / nl;
		}
	}
  
	for (i = 0; i < nm; i++)
	{
		for (j = 0; j < nl; j++)
		{
			D[i*nl + j] = ((DATA_TYPE) i*(j+2)) / nk;
		}
	}
}


// void compareResults(DATA_TYPE *G, DATA_TYPE *G_outputFromGpu)
// {
// 	int i,j,fail;
// 	fail = 0;

// 	for (i=0; i < ni; i++)
// 	{
// 		for (j=0; j < nl; j++)
// 		{
// 			if (percentDiff(G[i*nl + j], G_outputFromGpu[i*nl + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
// 			{
// 				fail++;				
// 			}
// 		}
// 	}
	
// 	// print results
// 	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
// }


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}

	
__global__ void mm3_kernel1(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *E, int ni, int nj, int nk, int nl, int nm)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < ni) && (j < nj))
	{
		int k;
		for(k=0; k < nk; k++)
		{
			E[i * nj + j] += A[i * nk + k] * B[k * nj + j];
		}
	}
}

	
__global__ void mm3_kernel2(DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *F, int ni, int nj, int nk, int nl, int nm)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < nj) && (j < nl))
	{
		int k;
		for(k=0; k < nm; k++)
		{
			F[i * nl + j] += C[i * nm + k] * D[k * nl +j];
		}
	}
}

	
__global__ void mm3_kernel3(DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G, int ni, int nj, int nk, int nl, int nm)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < ni) && (j < nl))
	{
		int k;
		for(k=0; k < nj; k++)
		{
			G[i * nl + j] += E[i * nj + k] * F[k * nl + j];
		}
	}
}


// void mm3_cpu(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G)
// {
// 	int i,j,k;
	
// 	/* E := A*B */
// 	for (i = 0; i < ni; i++)
// 	{
// 		for (j = 0; j < nj; j++)
// 		{
// 			E[i*nj + j] = 0;
// 			for (k = 0; k < nk; ++k)
// 			{
// 				E[i*nj + j] += A[i*nk + k] * B[k*nj + j];
// 			}
// 		}
// 	}
		
// 	/* F := C*D */
// 	for (i = 0; i < nj; i++)
// 	{
// 		for (j = 0; j < nl; j++)
// 		{
// 			F[i*nl + j] = 0;
// 			for (k = 0; k < nm; ++k)
// 			{
// 				F[i*nl + j] += C[i*nm + k] * D[k*nl + j];
// 			}
// 		}
// 	}

//   	/* G := E*F */
// 	for (i = 0; i < ni; i++)
// 	{
// 		for (j = 0; j < nl; j++)
// 		{
// 			G[i*nl + j] = 0;
// 			for (k = 0; k < nj; ++k)
// 			{
// 				G[i*nl + j] += E[i*nj + k] * F[k*nl + j];
// 			}
// 		}
// 	}
// }


void mm3Cuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E, DATA_TYPE* F, 
		DATA_TYPE* G, DATA_TYPE* G_outputFromGpu, int ni, int nj, int nk, int nl, int nm)
{
	double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;
	DATA_TYPE *D_gpu;
	DATA_TYPE *E_gpu;
	DATA_TYPE *F_gpu;
	DATA_TYPE *G_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * ni * nk);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * nk * nj);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * nj * nm);
	hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * nm * nl);
	hipMalloc((void **)&E_gpu, sizeof(DATA_TYPE) * ni * nj);
	hipMalloc((void **)&F_gpu, sizeof(DATA_TYPE) * nj * nl);
	hipMalloc((void **)&G_gpu, sizeof(DATA_TYPE) * ni * nl);

	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * ni * nk, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * nk * nj, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * nj * nm, hipMemcpyHostToDevice);
	hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * nm * nl, hipMemcpyHostToDevice);
	hipMemcpy(E_gpu, E, sizeof(DATA_TYPE) * ni * nj, hipMemcpyHostToDevice);
	hipMemcpy(F_gpu, F, sizeof(DATA_TYPE) * nj * nl, hipMemcpyHostToDevice);
	hipMemcpy(G_gpu, G, sizeof(DATA_TYPE) * ni * nl, hipMemcpyHostToDevice);	
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)nj) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)ni/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid2((size_t)(ceil( ((float)nl) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)nj/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid3((size_t)(ceil( ((float)nl) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)ni/ ((float)DIM_THREAD_BLOCK_Y) )));

	t_start = rtclock();
	mm3_kernel1<<<grid1,block>>>(A_gpu, B_gpu, E_gpu, ni, nj, nk, nl, nm);
	hipDeviceSynchronize();
	mm3_kernel2<<<grid2,block>>>(C_gpu, D_gpu, F_gpu, ni, nj, nk, nl, nm);
	hipDeviceSynchronize();
	mm3_kernel3<<<grid3,block>>>(E_gpu, F_gpu, G_gpu, ni, nj, nk, nl, nm);
	hipDeviceSynchronize();
	t_end = rtclock();
	hipMemcpy(G_outputFromGpu, G_gpu, sizeof(DATA_TYPE) * ni * nl, hipMemcpyDeviceToHost);

	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	hipFree(D_gpu);
	hipFree(E_gpu);
	hipFree(F_gpu);
	hipFree(G_gpu);
}


int main(int argc, char** argv)
{
	// double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;
	DATA_TYPE* C;
	DATA_TYPE* D;
	DATA_TYPE* E;
	DATA_TYPE* F;
	DATA_TYPE* G;
	DATA_TYPE* G_outputFromGpu;

	int ni, nj, nk, nl, nm;

	if(argc != 2){
		fprintf(stdout, "E.g.: exe size\n");
		return 1;
	}

	ni = atoi(argv[1]);
	nj = ni;
	nk = ni;
	nl = ni;
	nm = ni;

	A = (DATA_TYPE*)malloc(ni*nk*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(nk*nj*sizeof(DATA_TYPE));
	C = (DATA_TYPE*)malloc(nj*nm*sizeof(DATA_TYPE));
	D = (DATA_TYPE*)malloc(nm*nl*sizeof(DATA_TYPE));
	E = (DATA_TYPE*)malloc(ni*nj*sizeof(DATA_TYPE));
	F = (DATA_TYPE*)malloc(nj*nl*sizeof(DATA_TYPE));
	G = (DATA_TYPE*)malloc(ni*nl*sizeof(DATA_TYPE));
	G_outputFromGpu = (DATA_TYPE*)malloc(ni*nl*sizeof(DATA_TYPE));

	init_array(A, B, C, D, ni, nj, nk, nl, nm);

	GPU_argv_init();

	mm3Cuda(A, B, C, D, E, F, G, G_outputFromGpu, ni, nj, nk, nl, nm);

	// t_start = rtclock();

	// mm3_cpu(A, B, C, D, E, F, G);
	
	// t_end = rtclock();

	// fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	// compareResults(G, G_outputFromGpu);

	free(A);
	free(B);
	free(C);
	free(D);
	free(E);
	free(F);
	free(G);
	free(G_outputFromGpu);

	return 0;
}

